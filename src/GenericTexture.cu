#include "hip/hip_runtime.h"
#include "logging.h"
#include "cuda_utils.cuh"
#include "vector_utils.cuh"
#include "TextureImageLoader.h"
#include "GenericTexture.h"

void GenericTexture::copyToDevice(GenericTextureDevice* genericTextureDevice,
				  StatusCodes& status) const
{
  status = StatusCodes::NoError;

  GenericTextureDevice h_genericTextureDevice;

  h_genericTextureDevice.m_numVectors = m_vectors.size();
  h_genericTextureDevice.m_textureType = m_textureType;
  h_genericTextureDevice.m_textureObject = m_textureObject;

  // vectors
  int dataSize = m_vectors.size()*sizeof(float3);
  status = CCE(hipMalloc((void**)&(h_genericTextureDevice.m_vectors), dataSize));
  if (status != StatusCodes::NoError) {
    return;
  }
  status = CCE(hipMemcpy(h_genericTextureDevice.m_vectors, m_vectors.data(),
			  dataSize, hipMemcpyHostToDevice));
  if (status != StatusCodes::NoError) {
    return;
  }

  // whole texture
  status = CCE(hipMemcpy(genericTextureDevice, &h_genericTextureDevice,
			  sizeof(GenericTextureDevice), hipMemcpyHostToDevice));
  if (status != StatusCodes::NoError) {
    return;
  }
}


GenericTexture::GenericTexture(const pt::ptree& texture,
			       StatusCodes& status)
{
  std::string textureType = texture.get<std::string>("<xmlattr>.value");
  if (textureType == "SolidColor") {
    LOG_TRIVIAL(trace) << "Solid color texture.";
    m_textureType = TextureType::SolidColor;
    parseSolidColor(texture, status);
  }
  else if (textureType  == "ImageTexture") {
    LOG_TRIVIAL(trace) << "Image texture.";
    m_textureType = TextureType::ImageTexture;
    parseImageTexture(texture, status);
  }
}


void GenericTexture::parseSolidColor(const pt::ptree& texture,
				     StatusCodes& status)
{
  float3 albedo = string2float3(texture.get<std::string>("albedo.<xmlattr>.value"));
  m_vectors = {albedo};
}


void GenericTexture::parseImageTexture(const pt::ptree& texture,
				       StatusCodes& status)
{
  status = StatusCodes::NoError;
  int imageWidth;
  int imageHeight;
  int numChannels;
  std::vector<float4> h_imageBuffer;
  TextureImageLoader textureImageLoader(texture);
  textureImageLoader.loadImage(imageWidth, imageHeight, numChannels,
  			       h_imageBuffer, status);

  // create cuda texture
  // Allocate CUDA array in device memory
  hipChannelFormatDesc channelDesc =
    hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
  hipArray_t cuArray;
  status = CCE(hipMallocArray(&cuArray, &channelDesc,
			       imageWidth, imageHeight));
  if (status != StatusCodes::NoError) {
    return;
  }

  // Set pitch of the source (the width in memory in bytes
  // of the 2D array pointed to by src, including padding)
  const size_t spitch = imageWidth*sizeof(float4);
  // Copy data located at address h_data in host memory to device memory
  status = CCE(hipMemcpy2DToArray(cuArray, 0, 0, h_imageBuffer.data(), spitch,
				   imageWidth*sizeof(float4), imageHeight,
				   hipMemcpyHostToDevice));
  if (status != StatusCodes::NoError) {
    return;
  }

  // Specify texture
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = cuArray;

  // Specify texture object parameters
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 1;

  // Create texture object
  m_textureObject = 0;
  status = CCE(hipCreateTextureObject(&m_textureObject, &resDesc,
				       &texDesc, NULL));
  if (status != StatusCodes::NoError) {
    return;
  }
  if (m_textureObject == 0) {
    LOG_TRIVIAL(error) << "Could not create texture object.";
    status = StatusCodes::CudaError;
    return;
  }
}
