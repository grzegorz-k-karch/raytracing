#include "hip/hip_runtime.h"
#include <assert.h>

#include "logging.h"
#include "cuda_utils.cuh"
#include "Objects.cuh"
#include "SceneDevice.cuh"


__global__ void constructScene_kernel(
    const SceneRawObjectsDevice *sceneRawObjectsDevice,
    Camera* camera,
    Object** world)
{
  *camera = *(sceneRawObjectsDevice->camera);

  int numObjects = sceneRawObjectsDevice->numObjects;
  Object **objects = new Object*[numObjects];

  for (int objIdx = 0; objIdx < numObjects; objIdx++) {
    GenericMaterialDevice *genMatDev =
      &(sceneRawObjectsDevice->materials[objIdx]);
    Material *material = MaterialFactory::createMaterial(genMatDev);
    GenericObjectDevice *genObjDev =
      &(sceneRawObjectsDevice->objects[objIdx]);
    objects[objIdx] = ObjectFactory::createObject(genObjDev, material);
  }

  *world = createBVH(objects, numObjects);
}

void SceneDevice::constructScene(const SceneRawObjects& sceneRawObjects,
				 StatusCodes& status)
{
  status = StatusCodes::NoError;

  SceneRawObjectsDevice *d_sceneRawObjectsDevice =
    sceneRawObjects.getObjectsOnDevice(status);
  if (status != StatusCodes::NoError) {
    return;
  }

  // construct the scene on device
  status = CCE(hipMalloc((void**)&m_world, sizeof(Object*)));
  if (status != StatusCodes::NoError) {
    return;
  }
  status = CCE(hipMalloc((void**)&m_camera, sizeof(Camera)));
  if (status != StatusCodes::NoError) {
    return;
  }
  constructScene_kernel<<<1,1>>>(d_sceneRawObjectsDevice, m_camera, m_world);
  status = CCE(hipDeviceSynchronize());
  if (status != StatusCodes::NoError) {
    return;
  }
}
