#include "hip/hip_runtime.h"
#include "logging.h"
#include "cuda_utils.cuh"
#include "SceneDevice.cuh"

#include <assert.h>

__global__
void constructScene_kernel(const SceneRawObjectsDevice *sceneRawObjectsDevice,
			   Object** world)
{
  int numObjects = sceneRawObjectsDevice->numObjects;
  Object **objectList = new Object*[numObjects];

  for (int objIdx = 0; objIdx < numObjects; objIdx++) {
    GenericMaterialDevice *genMatDev =
      &(sceneRawObjectsDevice->materials[objIdx]);
    Material *mat = MaterialFactory::createMaterial(genMatDev);
    GenericObjectDevice *genObjDev =
      &(sceneRawObjectsDevice->objects[objIdx]);    
    objectList[objIdx] = ObjectFactory::createObject(genObjDev, mat);
  }
  *world = new ObjectList(objectList, numObjects);

  printf("GenericMaterialDevice destructor on device\n");
}

void SceneDevice::constructScene(const SceneRawObjectsDevice *sceneRawObjectsDevice,
				 StatusCodes& status)
{
  status = StatusCodes::NoError;
  
  // construct the scene on device
  status = CCE(hipMalloc((void**)&m_world, sizeof(Object*)));
  if (status != StatusCodes::NoError) {
    return;
  }
  constructScene_kernel<<<1,1>>>(sceneRawObjectsDevice, m_world);
  status = CCE(hipDeviceSynchronize());
  if (status != StatusCodes::NoError) {
    return;
  }
}
