#include "hip/hip_runtime.h"
#include <assert.h>

#include "logging.h"
#include "cuda_utils.cuh"
#include "Objects.cuh"
#include "SceneDevice.cuh"


__global__ void constructScene_kernel(
    const SceneRawObjectsDevice *sceneRawObjectsDevice,
    Camera* camera,
    Object** world)
{
  *camera = *(sceneRawObjectsDevice->m_camera);

  int numObjects = sceneRawObjectsDevice->m_numObjects;
  Object **objects = new Object*[numObjects];

  for (int objIdx = 0; objIdx < numObjects; objIdx++) {
    GenericObjectDevice *genObjDev =
      &(sceneRawObjectsDevice->m_objects[objIdx]);
    objects[objIdx] = ObjectFactory::createObject(genObjDev);
  }

  *world = createBVH(objects, numObjects);
}

void SceneDevice::constructScene(const SceneRawObjects& sceneRawObjects,
				 StatusCodes& status)
{
  status = StatusCodes::NoError;

  SceneRawObjectsDevice *d_sceneRawObjectsDevice =
    sceneRawObjects.getObjectsOnDevice(status);
  if (status != StatusCodes::NoError) {
    return;
  }

  // construct the scene on device
  status = CCE(hipMalloc((void**)&m_world, sizeof(Object*)));
  if (status != StatusCodes::NoError) {
    return;
  }
  status = CCE(hipMalloc((void**)&m_camera, sizeof(Camera)));
  if (status != StatusCodes::NoError) {
    return;
  }
  constructScene_kernel<<<1,1>>>(d_sceneRawObjectsDevice, m_camera, m_world);
  status = CCE(hipDeviceSynchronize());
  if (status != StatusCodes::NoError) {
    return;
  }
}
