#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

#include "cuda_utils.cuh"
#include "nvidia/hip/hip_vector_types.h"
#include "Renderer.cuh"


__global__
void initRandState_kernel(int imageWidth, int imageHeight, hiprandState* randState)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  if (i < imageWidth && j < imageHeight) {
    int pixelIdx = i + j*imageWidth;
    hiprand_init(1984, pixelIdx, 0, &randState[pixelIdx]);
  }
}

void Renderer::initRandState(StatusCodes &status)
{
  const int tx{8};
  const int ty{8};

  dim3 numThreads(tx, ty);
  dim3 numBlocks((m_imageWidth + tx - 1)/tx,
		 (m_imageHeight + ty - 1)/ty);

  status = CCE(hipMalloc((void**)&m_randState, m_imageWidth*m_imageHeight*sizeof(hiprandState)));
  if (status != StatusCodes::NoError) {
    return;
  }
  initRandState_kernel<<<numBlocks, numThreads>>>(m_imageWidth, m_imageHeight, m_randState);
  status = CCE(hipGetLastError());
  if (status != StatusCodes::NoError) {
    return;
  }
  status = CCE(hipDeviceSynchronize());
  if (status != StatusCodes::NoError) {
    return;
  }
}

void Renderer::initBuffers(StatusCodes &status)
{
  // initialize random state on device
  initRandState(status);
  if (status != StatusCodes::NoError) {
    return;
  }

  // allocate  buffer for the final image
  int framebufferSize = m_imageWidth*m_imageHeight*sizeof(float3);
  status = CCE(hipMallocManaged((void**)&m_framebuffer, framebufferSize));
  if (status != StatusCodes::NoError) {
    return;
  }
}

#define MY_FLOAT_MAX 3.402823e+38

__device__ float3 getBackgroundColor(const Ray& ray)
{
  float3 unitDirection = normalize(ray.m_direction);
  float t = 0.5f*(unitDirection.y + 1.0f);
  return (1.0f - t)*make_float3(1.0f, 1.0f, 1.0f) + t*make_float3(0.5f, 0.7f, 1.0f);
}

__device__ float3 getColor(const Ray& ray, Object* world,
			   hiprandState* localRandState)
{
  HitRecord hitRec;
  float3 color;
  Ray inRay = ray;
  float3 attenuationTotal = make_float3(1.0f, 1.0f, 1.0f);

  for (int i = 0; i < 50; i++) {
    if (world->hit(inRay, 0.001f, MY_FLOAT_MAX, hitRec)) {
      float3 attenuation;
      Ray scattered;
      if (hitRec.material->scatter(inRay, hitRec, attenuation,
      				   scattered, localRandState)) {
      	attenuationTotal *= attenuation;
      	inRay = scattered;
      }
    }
    else {
      color = getBackgroundColor(inRay);
      break;
    }
  }

  color *= attenuationTotal;

  return color;
}

__global__
void renderScene_kernel(Camera* camera, Object** world,
			hiprandState* randState, int imageWidth,
			int imageHeight, int sampleCount,
			float3* framebuffer)
{
  int pixelX = threadIdx.x + blockIdx.x*blockDim.x;
  int pixelY = threadIdx.y + blockIdx.y*blockDim.y;

  if (pixelX < imageWidth && pixelY < imageHeight) {

    int pixelIdx = pixelX + pixelY*imageWidth;
    hiprandState localRandState = randState[pixelIdx];
    float3 color = make_float3(0.0f, 0.0f, 0.0f);

    for (int sample = 0; sample < sampleCount; sample++) {
      float u = float(pixelX + hiprand_uniform(&localRandState))/float(imageWidth);
      float v = float(pixelY + hiprand_uniform(&localRandState))/float(imageHeight);
      Ray ray = camera->getRay(u, v, &localRandState);
      color += getColor(ray, *world, &localRandState);
    }
    framebuffer[pixelIdx] = color/float(sampleCount);
  }
}

void Renderer::renderScene(const SceneDevice &sceneDevice, StatusCodes &status)
{
  LOG_TRIVIAL(trace) << "Renderer::renderScene";
  const int tx = 8;
  const int ty = 8;

  dim3 numThreads(tx, ty);
  dim3 numBlocks((m_imageWidth + tx - 1)/tx,
		 (m_imageHeight + ty - 1)/ty);

  renderScene_kernel<<<numBlocks, numThreads>>>(sceneDevice.m_camera,
						sceneDevice.m_world,
						m_randState, m_imageWidth,
						m_imageHeight, m_sampleCount,
						m_framebuffer);
  status = CCE(hipDeviceSynchronize());
  if (status != StatusCodes::NoError) {
    return;
  }
}

void Renderer::getImageOnHost(std::vector<float3>& image, StatusCodes& status) const
{
  int imageSize = m_imageWidth*m_imageHeight;
  image.resize(imageSize);

  for (int pixelIdx = 0; pixelIdx < imageSize; pixelIdx++) {
    image[pixelIdx] = m_framebuffer[pixelIdx];
  }
}
