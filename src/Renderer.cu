#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

#include "cuda_utils.cuh"
#include "vector_utils.h"
#include "Renderer.cuh"


__global__
void initRandState_kernel(int imageWidth, int imageHeight, hiprandState* randState)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  if (i < imageWidth && j < imageHeight) {
    int pixelIdx = i + j*imageWidth;
    hiprand_init(1984, pixelIdx, 0, &randState[pixelIdx]);
  }
}

void Renderer::initRandState(StatusCodes &status)
{
  const int tx{8};
  const int ty{8};

  dim3 numThreads(tx, ty);
  dim3 numBlocks((m_imageWidth + tx - 1)/tx,
		 (m_imageHeight + ty - 1)/ty);

  status = CCE(hipMalloc((void**)&m_randState, m_imageWidth*m_imageHeight*sizeof(hiprandState)));
  if (status != StatusCodes::NoError) {
    return;
  }
  initRandState_kernel<<<numBlocks, numThreads>>>(m_imageWidth, m_imageHeight, m_randState);
  status = CCE(hipGetLastError());
  if (status != StatusCodes::NoError) {
    return;
  }
  status = CCE(hipDeviceSynchronize());
  if (status != StatusCodes::NoError) {
    return;
  }
}

void Renderer::initBuffers(StatusCodes &status)
{
  // initialize random state on device
  initRandState(status);
  if (status != StatusCodes::NoError) {
    return;
  }

  // allocate  buffer for the final image
  int framebufferSize = m_imageWidth*m_imageHeight*sizeof(float3);
  status = CCE(hipMallocManaged((void**)&m_framebuffer, framebufferSize));
  if (status != StatusCodes::NoError) {
    return;
  }
}

__global__
void renderScene_kernel(Camera* camera, Object** world,
			hiprandState* randState, int imageWidth,
			int imageHeight, int sampleCount,
			float3* framebuffer)
{
  int pixelX = threadIdx.x + blockIdx.x*blockDim.x;
  int pixelY = threadIdx.y + blockIdx.y*blockDim.y;
  
  if (pixelX < imageWidth && pixelY < imageHeight) {

    int pixelIdx = pixelX + pixelY*imageWidth;
    hiprandState localRandState = randState[pixelIdx];
    float3 color = make_float3(0.0f, 0.0f, 0.0f);
    
    for (int sample = 0; sample < sampleCount; sample++) {
      float u = float(pixelX +
		      hiprand_uniform(&localRandState))/float(imageWidth);
      float v = float(pixelY +
		      hiprand_uniform(&localRandState))/float(imageHeight);
      Ray ray = camera->getRay(u, v, &localRandState);
      // color += getColor(ray, *world, &localRandState);
    }
    framebuffer[pixelIdx] = color/float(sampleCount);
  }
}

void Renderer::renderScene(const SceneDevice &sceneDevice,
                           StatusCodes &status)
{
  const int tx = 8;
  const int ty = 8;

  dim3 numThreads(tx, ty);
  dim3 numBlocks((m_imageWidth + tx - 1)/tx,
		 (m_imageHeight + ty - 1)/ty);

  renderScene_kernel<<<numBlocks, numThreads>>>(sceneDevice.m_camera,
						sceneDevice.m_world,
						m_randState, m_imageWidth,
						m_imageHeight, m_sampleCount,
						m_framebuffer);
  status = CCE(hipDeviceSynchronize());
  if (status != StatusCodes::NoError) {
    return;
  }
}

void Renderer::getImageOnHost(std::vector<float3>& image, StatusCodes& status) const
{
  int imageSize = m_imageWidth*m_imageHeight;
  image.resize(imageSize);

  for (int pixelIdx = 0; pixelIdx < imageSize; pixelIdx++) {
    image[pixelIdx] = m_framebuffer[pixelIdx];
  }
}
