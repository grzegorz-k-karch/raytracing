#define _USE_MATH_DEFINES
#include <cmath>

#include "Camera.cuh"
#include "nvidia/hip/hip_vector_types.h"
#include "cuda_utils.cuh"
#include "vector_utils.cuh"

void Camera::copyToDevice(Camera* cameraDevice, StatusCodes& status) const
{
  status = StatusCodes::NoError;

  status = CCE(hipMemcpy(cameraDevice, this,
			  sizeof(Camera), hipMemcpyHostToDevice));
}

