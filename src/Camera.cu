#define _USE_MATH_DEFINES
#include <cmath>

#include "Camera.cuh"
#include "nvidia/hip/hip_vector_types.h"
#include "cuda_utils.cuh"
#include "vector_utils.cuh"

void Camera::copyToDevice(Camera* d_camera, StatusCodes& status) const
{
  status = StatusCodes::NoError;

  status = CCE(hipMemcpy(d_camera, this,
			  sizeof(Camera), hipMemcpyHostToDevice));
}

