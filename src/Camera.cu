#define _USE_MATH_DEFINES
#include <cmath>

#include "Camera.cuh"
#include "nvidia/hip/hip_vector_types.h"
#include "cuda_utils.cuh"
#include "vector_utils.cuh"

void Camera::copyToDevice(Camera* d_camera, StatusCode& status) const
{
  status = StatusCode::NoError;

  status = CCE(hipMemcpy(d_camera, this,
			  sizeof(Camera), hipMemcpyHostToDevice));
}

