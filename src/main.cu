#include "hip/hip_runtime.h"
#include "gkk_cuda_utils.cuh"
#include "gkk_vec.cuh"
#include "gkk_color.cuh"
#include "gkk_object.cuh"
#include "gkk_geometry.cuh"
#include "gkk_camera.cuh"

#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <fstream>
#include <string>
#include <boost/program_options.hpp>
namespace po = boost::program_options;

int load_obj(std::string input,
	     std::vector<vec3>& point_list,
	     std::vector<int>& triangle_list)
{
  std::fstream fs(input, std::fstream::in);
  char c;
  float x, y, z;
  int prev_pos = 0;
  while (fs >> c >> x >> y >> z) {
    if (c != 'v') {
      fs.seekg(prev_pos, fs.beg);
      break;
    }
    prev_pos = fs.tellg();
    vec3 v(x, y, z);
    point_list.push_back(v);
  }

  int i0, i1, i2;
  while (fs >> c >> i0 >> i1 >> i2) {
    if (c != 'f') {
      break;
    }
    triangle_list.push_back(i0-1);
    triangle_list.push_back(i1-1);
    triangle_list.push_back(i2-1);
  }

  fs.close();
  return 0;
}

// assuming pixel values are in range (0,1)
int write_ppm(vec3* raw_image,
	      const int nx,
	      const int ny,
	      std::string output)
{
  std::fstream fs(output, std::fstream::out);
  fs << "P3\n" << nx << " " << ny << "\n255\n";
  for (int j = ny-1; j >= 0; j--) {
    for (int i = 0; i < nx; i++) {
      size_t pixel_idx = i + j*nx;
      vec3 color = raw_image[pixel_idx];
      // gamma correction
      color = vec3(std::sqrt(color.r()), std::sqrt(color.g()), std::sqrt(color.b()));
      int ir = int(255.99f*color.r());
      int ig = int(255.99f*color.g());
      int ib = int(255.99f*color.b());
      fs << ir << " " << ig << " " << ib << "\n";
    }
  }
  fs.close();
  return 0;
}


__global__ void render_init(int max_x, int max_y, hiprandState* rand_state)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  if ((i >= max_x) || (j >= max_y)) {
    return;
  }

  int pixel_idx = i + j*max_x;
  hiprand_init(1984, pixel_idx, 0, &rand_state[pixel_idx]);
}


__global__ void render(vec3* fb, int max_x, int max_y, int ns,
		       Object** world, hiprandState* rand_state)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  if ((i >= max_x) || (j >= max_y)) {
    return;
  }

  int pixel_idx = i + j*max_x;
  hiprandState local_rand_state = rand_state[pixel_idx];
  vec3 color = vec3(0.0f, 0.0f, 0.0f);

  // default camera
  vec3 lookfrom = vec3(-1.0f, 2.7f, 4.5f);
  vec3 lookat = vec3(0.0f, 1.0f, 0.0f);
  vec3 up = vec3(0.0f, 1.0f, 0.0f);
  Camera camera(lookfrom, lookat, up, 60.0f, float(max_x)/float(max_y),
		0.125f, (lookfrom-lookat).length());

  for (int s = 0; s < ns; s++) {
    float u = float(i + hiprand_uniform(&local_rand_state))/float(max_x);
    float v = float(j + hiprand_uniform(&local_rand_state))/float(max_y);
    Ray ray = camera.get_ray(u, v, &local_rand_state);
    color += get_color(ray, *world, &local_rand_state);
  }
  fb[pixel_idx] = color/float(ns);
}


__global__ void create_world(Object** obj_list, Object** world, int n,
			     hiprandState* rand_state,
			     vec3* point_list, int num_points,
			     int* triangle_list, int num_triangles)
{
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    hiprandState local_rand_state = rand_state[0];

    obj_list[0] = new Sphere(vec3(0.0f, -1000.0f, 0.0f), 1000.0f,
			     new Lambertian(vec3(0.5f, 0.5f, 0.5f)));

    obj_list[1] = new Sphere(vec3(0.0f, 1.0f, 0.0f), 1.0f, new Dielectric(1.5f));
    obj_list[2] = new Sphere(vec3(-4.0f, 1.0f, 0.0f), 1.0f, new Lambertian(vec3(0.4f, 0.2f, 0.1f)));
    obj_list[3] = new Sphere(vec3(4.0f, 1.0f, 0.0f), 1.0f, new Metal(vec3(0.7f, 0.6f, 0.5f), 0.0f));

    obj_list[4] = new TriangleMesh(point_list, num_points, triangle_list, num_triangles,
				   new Metal(vec3(1.0f, 0.6f, 0.5f), 0.0f));

    int i = 5;
    for (int a = -11; a < 11; a++) {
      for (int b = -11; b < 11; b++) {
    	float choose_mat = hiprand_uniform(&local_rand_state);
    	vec3 center(a + 0.9f*hiprand_uniform(&local_rand_state),
    		    0.2f,
    		    b + 0.9f*hiprand_uniform(&local_rand_state));
    	if ((center - vec3(4.0f, 0.2f, 0.0f)).length() > 0.9f) {
    	  if (choose_mat < 0.5f) { // diffuse
    	    obj_list[i++] =
    	      new Sphere(center, 0.2f,
    			 new Lambertian(vec3(hiprand_uniform(&local_rand_state)*hiprand_uniform(&local_rand_state),
    					     hiprand_uniform(&local_rand_state)*hiprand_uniform(&local_rand_state),
    					     hiprand_uniform(&local_rand_state)*hiprand_uniform(&local_rand_state))));
    	  }
    	  else if (choose_mat < 0.75f) { // metal
    	    obj_list[i++] =
    	      new Sphere(center, 0.2f,
    			 new Metal(vec3(0.5f*(1.0f + hiprand_uniform(&local_rand_state)),
    					0.5f*(1.0f + hiprand_uniform(&local_rand_state)),
    					0.5f*(1.0f + hiprand_uniform(&local_rand_state))),
    				   0.5f*hiprand_uniform(&local_rand_state)));
    	  }
    	  else { // glass
    	    obj_list[i++] = new Sphere(center, 0.2f, new Dielectric(1.5f));
    	  }
    	}
    	if (i >= n) {
    	  break;
    	}
      }
      if (i >= n) {
    	break;
      }
    }
    *world = new ObjectList(obj_list, n);
  }
}


__global__ void free_world(Object** d_list, Object** d_world, int n)
{
  for (int i = 0; i < n; i++) {
    delete d_list[i];
  }
  delete *d_world;
}


void generate_test_image(vec3* raw_image,
			 const int nx, const int ny,
			 const int num_samples)
{

  int tx = 8;
  int ty = 8;

  dim3 blocks((nx+tx-1)/tx, (ny+ty-1)/ty);
  dim3 threads(tx, ty);

  hiprandState *d_rand_state;
  checkCudaErrors(hipMalloc((void**)&d_rand_state, nx*ny*sizeof(hiprandState)));

  render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  std::vector<vec3> point_list;
  // point_list.push_back(vec3(0.0f, 0.0f, 0.0f));
  // point_list.push_back(vec3(3.0f, 0.0f, 0.0f));
  // point_list.push_back(vec3(0.0f, 3.0f, 0.0f));
  // point_list.push_back(vec3(0.0f, 0.0f, 3.0f));
  std::vector<int> triangle_list;
  // triangle_list.push_back(0);
  // triangle_list.push_back(1);
  // triangle_list.push_back(2);
  // triangle_list.push_back(0);
  // triangle_list.push_back(2);
  // triangle_list.push_back(3);
  // triangle_list.push_back(0);
  // triangle_list.push_back(3);
  // triangle_list.push_back(1);
  // triangle_list.push_back(1);
  // triangle_list.push_back(3);
  // triangle_list.push_back(2);

  load_obj("../models/teapot.obj", point_list, triangle_list);

  int num_triangles = triangle_list.size()/3;
  int num_points = point_list.size();

  vec3 *d_point_list;
  checkCudaErrors(hipMalloc((void**)&d_point_list, num_points*sizeof(vec3)));
  int *d_triangle_list;
  checkCudaErrors(hipMalloc((void**)&d_triangle_list, num_triangles*3*sizeof(int)));

  checkCudaErrors(hipMemcpy(d_point_list, point_list.data(), num_points*sizeof(vec3), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_triangle_list, triangle_list.data(),
			     num_triangles*3*sizeof(int), hipMemcpyHostToDevice));

  int num_spheres = 480;  

  Object **d_list;
  checkCudaErrors(hipMalloc((void**)&d_list, (num_spheres + 1)*sizeof(Object*)));
  Object **d_world;
  checkCudaErrors(hipMalloc((void**)&d_world, sizeof(Object*)));

  create_world<<<1, 1>>>(d_list, d_world, num_spheres + 1, d_rand_state,
			 d_point_list, num_points, d_triangle_list, num_triangles);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  render<<<blocks, threads>>>(raw_image, nx, ny, num_samples, d_world, d_rand_state);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  free_world<<<1, 1>>>(d_list, d_world, num_spheres + 1);
  checkCudaErrors(hipGetLastError());

  checkCudaErrors(hipFree(d_point_list));
  checkCudaErrors(hipFree(d_triangle_list));
  checkCudaErrors(hipFree(d_list));
  checkCudaErrors(hipFree(d_world));
  checkCudaErrors(hipFree(d_rand_state));
}


int main(int argc, char** argv)
{
  std::string output = "";
  int nx;
  int ny;
  int ns;
  
  try {
    po::options_description desc{"Options"};
    desc.add_options()
      ("help,h", "Help screen")
      ("output,o", po::value<std::string>(&output)->required(), "Filename for the output figure")
      ("resolution-x,x", po::value<int>(&nx)->default_value(1600), "Horizontal output resolution")
      ("resolution-y,y", po::value<int>(&ny)->default_value(800), "Vertical output resolution")
      ("num-samples,s", po::value<int>(&ns)->default_value(100), "Number of samples per pixel");
    po::variables_map vm;
    po::store(po::parse_command_line(argc, argv, desc), vm);

    if (vm.count("help")) {
      std::cout << desc << std::endl;
      return 0;
    }
    po::notify(vm);
  }
  catch(const std::runtime_error &ex) {
    std::cerr << ex.what() << std::endl;
  }

  int num_pixels = nx*ny;

  size_t fb_size = num_pixels*sizeof(vec3);
  vec3 *fb;
  checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

  generate_test_image(fb, nx, ny, ns);
  write_ppm(fb, nx, ny, output);

  hipFree(fb);

  return 0;
}
