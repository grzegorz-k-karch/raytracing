#include "hip/hip_runtime.h"
#include "gkk_cuda_utils.cuh"
#include "gkk_vec.cuh"
#include "gkk_color.cuh"
#include "gkk_object.cuh"
#include "gkk_geometry.cuh"
#include "gkk_camera.cuh"

#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <fstream>
#include <string>
#include <boost/program_options.hpp>
namespace po = boost::program_options;


// assuming pixel values are in range (0,1)
int write_ppm(vec3* raw_image,
	      const int nx,
	      const int ny,
	      std::string output)
{
  std::fstream fs(output, std::fstream::out);
  fs << "P3\n" << nx << " " << ny << "\n255\n";
  for (int j = ny-1; j >= 0; j--) {
    for (int i = 0; i < nx; i++) {
      size_t pixel_idx = i + j*nx;
      vec3 color = raw_image[pixel_idx];
      // gamma correction
      color = vec3(std::sqrt(color.r()), std::sqrt(color.g()), std::sqrt(color.b()));
      int ir = int(255.99f*color.r());
      int ig = int(255.99f*color.g());
      int ib = int(255.99f*color.b());
      fs << ir << " " << ig << " " << ib << "\n";
    }
  }
  fs.close();
  return 0;
}


__global__ void render_init(int max_x, int max_y, hiprandState* rand_state)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  if ((i >= max_x) || (j >= max_y)) {
    return;
  }

  int pixel_idx = i + j*max_x;
  hiprand_init(1984, pixel_idx, 0, &rand_state[pixel_idx]);
}


__global__ void render(vec3* fb, int max_x, int max_y, int ns,
		       Object** world, hiprandState* rand_state)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;
  if ((i >= max_x) || (j >= max_y)) {
    return;
  }

  int pixel_idx = i + j*max_x;
  hiprandState local_rand_state = rand_state[pixel_idx];
  vec3 color = vec3(0.0f, 0.0f, 0.0f);

  // default camera
  vec3 lookfrom = vec3(0.0f, 2.2f, -5.0f);
  vec3 lookat = vec3(0.0f, 0.0f, 0.0f);
  vec3 up = vec3(0.0f, 1.0f, 0.0f);
  Camera camera(lookfrom, lookat, up, 60.0f, float(max_x)/float(max_y),
		0.125f, (lookfrom-lookat).length());

  for (int s = 0; s < ns; s++) {
    float u = float(i + hiprand_uniform(&local_rand_state))/float(max_x);
    float v = float(j + hiprand_uniform(&local_rand_state))/float(max_y);
    Ray ray = camera.get_ray(u, v, &local_rand_state);
    color += get_color(ray, *world, &local_rand_state);
  }
  fb[pixel_idx] = color/float(ns);
}


__global__ void create_world(Object** d_list, Object** d_world, int n,
			     hiprandState* rand_state)
{
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    hiprandState local_rand_state = rand_state[0];

    d_list[0] = new Sphere(vec3(0.0f, -1000.0f, 0.0f), 1000.0f,
    			   new Lambertian(vec3(0.5f, 0.5f, 0.5f)));

    d_list[1] = new Sphere(vec3(0.0f, 1.0f, 0.0f), 1.0f, new Dielectric(1.5f));
    d_list[2] = new Sphere(vec3(-4.0f, 1.0f, 0.0f), 1.0f, new Lambertian(vec3(0.4f, 0.2f, 0.1f)));
    d_list[3] = new Sphere(vec3(4.0f, 1.0f, 0.0f), 1.0f, new Metal(vec3(0.7f, 0.6f, 0.5f), 0.0f));

    int i = 4;
    for (int a = -11; a < 11; a++) {
      for (int b = -11; b < 11; b++) {
    	float choose_mat = hiprand_uniform(&local_rand_state);
    	vec3 center(a + 0.9f*hiprand_uniform(&local_rand_state),
    		    0.2f,
    		    b + 0.9f*hiprand_uniform(&local_rand_state));
    	if ((center - vec3(4.0f, 0.2f, 0.0f)).length() > 0.9f) {
    	  if (choose_mat < 0.5f) { // diffuse
    	    d_list[i++] =
    	      new Sphere(center, 0.2f,
    			 new Lambertian(vec3(hiprand_uniform(&local_rand_state)*hiprand_uniform(&local_rand_state),
    					     hiprand_uniform(&local_rand_state)*hiprand_uniform(&local_rand_state),
    					     hiprand_uniform(&local_rand_state)*hiprand_uniform(&local_rand_state))));
    	  }
    	  else if (choose_mat < 0.75f) { // metal
    	    d_list[i++] =
    	      new Sphere(center, 0.2f,
    			 new Metal(vec3(0.5f*(1.0f + hiprand_uniform(&local_rand_state)),
    					0.5f*(1.0f + hiprand_uniform(&local_rand_state)),
    					0.5f*(1.0f + hiprand_uniform(&local_rand_state))),
    				   0.5f*hiprand_uniform(&local_rand_state)));
    	  }
    	  else { // glass
    	    d_list[i++] = new Sphere(center, 0.2f, new Dielectric(1.5f));
    	  }
    	}
	if (i >= n) {
	  break;
	}
      }
      if (i >= n) {
	break;
      }
    }
    *d_world = new ObjectList(d_list, n);
  }
}


__global__ void free_world(Object** d_list, Object** d_world, int n)
{
  for (int i = 0; i < n; i++) {
    delete d_list[i];
  }
  delete *d_world;
}


void generate_test_image(vec3* raw_image,
			 const int nx, const int ny,
			 const int num_samples)
{

  int tx = 8;
  int ty = 8;

  dim3 blocks((nx+tx-1)/tx, (ny+ty-1)/ty);
  dim3 threads(tx, ty);

  hiprandState *d_rand_state;
  checkCudaErrors(hipMalloc((void**)&d_rand_state, nx*ny*sizeof(hiprandState)));

  render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  int num_spheres = 480;

  Object **d_list;
  checkCudaErrors(hipMalloc((void**)&d_list, num_spheres*sizeof(Object*)));
  Object **d_world;
  checkCudaErrors(hipMalloc((void**)&d_world, sizeof(Object*)));

  create_world<<<1, 1>>>(d_list, d_world, num_spheres, d_rand_state);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  render<<<blocks, threads>>>(raw_image, nx, ny, num_samples, d_world, d_rand_state);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  free_world<<<1, 1>>>(d_list, d_world, num_spheres);
  checkCudaErrors(hipGetLastError());

  checkCudaErrors(hipFree(d_list));
  checkCudaErrors(hipFree(d_world));
  checkCudaErrors(hipFree(d_rand_state));
}


int main(int argc, char** argv)
{
  std::string output = "";
  int nx;
  int ny;
  int ns;

  try {
    po::options_description desc{"Options"};
    desc.add_options()
      ("help,h", "Help screen")
      ("output,o", po::value<std::string>(&output)->required(), "Filename for the output figure")
      ("resolution-x,x", po::value<int>(&nx)->default_value(1600), "Horizontal output resolution")
      ("resolution-y,y", po::value<int>(&ny)->default_value(800), "Vertical output resolution")
      ("num-samples,s", po::value<int>(&ns)->default_value(100), "Number of samples per pixel");
    po::variables_map vm;
    po::store(po::parse_command_line(argc, argv, desc), vm);

    if (vm.count("help")) {
      std::cout << desc << std::endl;
      return 0;
    }
    po::notify(vm);
  }
  catch(const std::runtime_error &ex) {
    std::cerr << ex.what() << std::endl;
  }

  int num_pixels = nx*ny;

  size_t fb_size = num_pixels*sizeof(vec3);
  vec3 *fb;
  checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

  generate_test_image(fb, nx, ny, ns);
  write_ppm(fb, nx, ny, output);

  hipFree(fb);

  return 0;
}
