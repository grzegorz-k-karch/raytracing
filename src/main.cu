#include "hip/hip_runtime.h"
#include "gkk_cuda_utils.cuh"
#include "gkk_vec.cuh"
#include "gkk_color.cuh"
#include "gkk_object.cuh"
#include "gkk_geometry.cuh"

#include <iostream>

#include <hiprand/hiprand_kernel.h>

// assuming pixel values are in range (0,1)
int write_ppm(vec3* raw_image,
	      const int nx=300,
	      const int ny=200)
{
  std::cout << "P3\n" << nx << " " << ny << "\n255\n";
  for (int j = ny-1; j >= 0; j--) {
    for (int i = 0; i < nx; i++) {
      size_t pixel_idx = i + j*nx;
      vec3 color = raw_image[pixel_idx];
      // gamma correction
      color = vec3(std::sqrt(color.r()), std::sqrt(color.g()), std::sqrt(color.b()));
      int ir = int(255.99f*color.r());
      int ig = int(255.99f*color.g());
      int ib = int(255.99f*color.b());
      std::cout << ir << " " << ig << " " << ib << "\n";
    }
  }
  return 0;
}


__device__ bool hit_sphere(const vec3& center, float radius, const Ray& ray)
{
  vec3 oc = ray.o - center;
  float a = dot(ray.d, ray.d);
  float b = 2.0f*dot(oc, ray.d);
  float c = dot(oc, oc) - radius*radius;
  float discriminant = b*b - 4*a*c;

  return (discriminant > 0.0f);
}


__global__ void render(vec3* fb, int max_x, int max_y,
		       vec3 lower_left_corner, vec3 horizontal,
		       vec3 vertical, vec3 origin)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;

  if ((i >= max_x) || (j >= max_y)) {
    return;
  }

  int pixel_idx = i + j*max_x;
  float u = float(i)/float(max_x);
  float v = float(j)/float(max_y);
  Ray ray(origin, lower_left_corner + u*horizontal + v*vertical);
  vec3 color = get_plane_color(ray);
  vec3 sphere_center = vec3(0.0f, 0.0f, -2.0f);
  float sphere_radius = 0.5f;
  if (hit_sphere(sphere_center, sphere_radius, ray)) {
    color = vec3(1.0f, 0.0f, 0.0f);
  }
  fb[pixel_idx] = color;
}


__global__ void render(vec3* fb, int max_x, int max_y,
		       vec3 lower_left_corner, vec3 horizontal,
		       vec3 vertical, vec3 origin, Object** world)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int j = threadIdx.y + blockIdx.y*blockDim.y;

  if ((i >= max_x) || (j >= max_y)) {
    return;
  }

  int pixel_idx = i + j*max_x;
  float u = float(i)/float(max_x);
  float v = float(j)/float(max_y);
  Ray ray(origin, lower_left_corner + u*horizontal + v*vertical);
  vec3 color = get_color(ray, *world); //, 0);
  fb[pixel_idx] = color;
}


__global__ void create_world(Object** d_list, Object** d_world)
{
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    *(d_list) = new Sphere(vec3(0.0f, 0.0f, -1.0f), 0.5f,
			   new Lambertian(vec3(0.5f, 0.5f, 0.5f)));
    *(d_list + 1) = new Sphere(vec3(0.0f, -100.5f, -1.0f), 100.0f,
			       new Lambertian(vec3(0.5f, 0.5f, 0.5f)));
    *d_world = new ObjectList(d_list, 2);
  }
}


__global__ void free_world(Object** d_list, Object** d_world)
{
  delete *(d_list);
  delete *(d_list + 1);
  delete *d_world;
}


void generate_test_image(vec3* raw_image,
			const int nx=400,
			const int ny=200)
{

  int tx = 8;
  int ty = 8;

  dim3 blocks((nx+tx-1)/tx, (ny+ty-1)/ty);
  dim3 threads(tx, ty);

  Object **d_list;
  checkCudaErrors(hipMalloc((void**)&d_list, 2*sizeof(Object*)));
  Object **d_world;
  checkCudaErrors(hipMalloc((void**)&d_world, sizeof(Object*)));

  create_world<<<1, 1>>>(d_list, d_world);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  render<<<blocks, threads>>>(raw_image, nx, ny, vec3(-2.0f, -1.0f, -1.0f),
			      vec3(4.0f, 0.0f, 0.0f), vec3(0.0, 2.0f, 0.0f),
			      vec3(0.0f, 0.0f, 0.0f), d_world);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  free_world<<<1, 1>>>(d_list, d_world);
  checkCudaErrors(hipGetLastError());
  
  checkCudaErrors(hipFree(d_list));
  checkCudaErrors(hipFree(d_world));
}

int main()
{
  int nx = 1600;
  int ny = 800;

  int num_pixels = nx*ny;

  size_t fb_size = num_pixels*sizeof(vec3);
  vec3 *fb;
  checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

  generate_test_image(fb, nx, ny);
  write_ppm(fb, nx, ny);

  hipFree(fb);

  return 0;
}
