#include "hip/hip_runtime.h"
#include "vector_utils.cuh"


__device__
float3 randomInUnitSphere(hiprandState* localRandState)
{
  float3 p;
  do {
    p = 2.0f*make_float3(hiprand_uniform(localRandState),
			 hiprand_uniform(localRandState),
			 hiprand_uniform(localRandState)) - make_float3(1.0f, 1.0f, 1.0f);
    
  } while (squaredLength(p) >= 1.0f);
  return p;
}


__device__
float3 randomInUnitDisk(hiprandState* localRandState)
{
  float3 p;
  do {
    p = 2.0f*make_float3(hiprand_uniform(localRandState),
			 hiprand_uniform(localRandState),
			 0.0f) - make_float3(1.0f, 1.0f, 0.0f);
    
  } while (squaredLength(p) >= 1.0f);
  return p;  
}

