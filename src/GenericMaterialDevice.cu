#include "hip/hip_runtime.h"
#include "GenericMaterial.h"

void GenericMaterial::copyToDevice(GenericMaterialDevice* genericMaterialDevice)
{
  GenericMaterialDevice *h_genericMaterial = new GenericMaterialDevice;

  // scalars
  int dataSize = m_scalars.size()*sizeof(float);
  CCE(hipMalloc((void**)&(h_genericMaterial->scalars), dataSize));
  CCE(hipMemcpy(h_genericMaterial->scalars, m_scalars.data(),
	     dataSize, hipMemcpyHostToDevice);
  // vectors
  dataSize = m_vectors.size()*sizeof(float3);
  CCE(hipMalloc((void**)&(h_genericMaterial->vectors), dataSize));
  CCE(hipMemcpy(h_genericMaterial->vectors, m_vectors.data(),
	     dataSize, hipMemcpyHostToDevice);

  h_genericMaterial->numScalars = m_scalars.size();
  h_genericMaterial->numVectors = m_vectors.size();
  
  // whole material
  CCE(hipMemcpy(genericMaterialDevice, h_genericMaterial,
	     sizeof(GenericMaterialDevice), hipMemcpyHostToDevice);
  
  delete h_genericMaterial;
}

