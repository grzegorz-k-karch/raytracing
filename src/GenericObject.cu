#include "hip/hip_runtime.h"
#include "logging.h"
#include "cuda_utils.cuh"
#include "GenericObject.h"
#include "GenericMaterial.h"

void GenericObject::copyToDevice(GenericObjectDevice* genericObjectDevice,
				 StatusCodes& status)
{
  status = StatusCodes::NoError;

  // copy all sizes and object type ------------------------------------------
  m_h_genericObjectDevice.m_bmin = m_bbox.min();
  m_h_genericObjectDevice.m_bmax = m_bbox.max();
  m_h_genericObjectDevice.m_numScalars = m_scalars.size();
  m_h_genericObjectDevice.m_numVectors = m_vectors.size();
  m_h_genericObjectDevice.m_numVertices = m_vertices.size();
  m_h_genericObjectDevice.m_numVertexColors = m_vertexColors.size();
  m_h_genericObjectDevice.m_numVertexNormals = m_vertexNormals.size();
  m_h_genericObjectDevice.m_numTextureCoords = m_textureCoords.size();
  m_h_genericObjectDevice.m_numTriangleIndices = m_triangleIndices.size();
  m_h_genericObjectDevice.m_objectType = m_objectType;

  // material ----------------------------------------------------------------
  // allocate buffer for GenericMaterialDevice struct
  int dataSize = sizeof(GenericMaterialDevice);
  status = CCE(hipMalloc((void**)&(m_h_genericObjectDevice.m_material),
			  dataSize));
  if (status != StatusCodes::NoError) {
    return;
  }
  m_material->copyToDevice(m_h_genericObjectDevice.m_material, status);
  if (status != StatusCodes::NoError) {
    return;
  }

  // scalars -----------------------------------------------------------------
  dataSize = m_scalars.size()*sizeof(float);
  status = CCE(hipMalloc((void**)&(m_h_genericObjectDevice.m_scalars), dataSize));
  if (status != StatusCodes::NoError) {
    return;
  }
  status = CCE(hipMemcpy(m_h_genericObjectDevice.m_scalars, m_scalars.data(),
			  dataSize, hipMemcpyHostToDevice));
  if (status != StatusCodes::NoError) {
    return;
  }

  // vectors -----------------------------------------------------------------
  dataSize = m_vectors.size()*sizeof(float3);
  status = CCE(hipMalloc((void**)&(m_h_genericObjectDevice.m_vectors), dataSize));
  if (status != StatusCodes::NoError) {
    return;
  }
  status = CCE(hipMemcpy(m_h_genericObjectDevice.m_vectors, m_vectors.data(),
			  dataSize, hipMemcpyHostToDevice));
  if (status != StatusCodes::NoError) {
    return;
  }

  // vertices ----------------------------------------------------------------
  dataSize = m_vertices.size()*sizeof(float3);
  status = CCE(hipMalloc((void**)&(m_h_genericObjectDevice.m_vertices),
			  dataSize));
  if (status != StatusCodes::NoError) {
    return;
  }
  status = CCE(hipMemcpy(m_h_genericObjectDevice.m_vertices, m_vertices.data(),
			  dataSize, hipMemcpyHostToDevice));
  if (status != StatusCodes::NoError) {
    return;
  }

  // vertex colors -----------------------------------------------------------
  dataSize = m_vertexColors.size()*sizeof(float3);
  status = CCE(hipMalloc((void**)&(m_h_genericObjectDevice.m_vertexColors),
			  dataSize));
  if (status != StatusCodes::NoError) {
    return;
  }
  status = CCE(hipMemcpy(m_h_genericObjectDevice.m_vertexColors,
			  m_vertexColors.data(), dataSize,
			  hipMemcpyHostToDevice));
  if (status != StatusCodes::NoError) {
    return;
  }

  // vertex normals ----------------------------------------------------------
  dataSize = m_vertexNormals.size()*sizeof(float3);
  status = CCE(hipMalloc((void**)&(m_h_genericObjectDevice.m_vertexNormals),
			  dataSize));
  if (status != StatusCodes::NoError) {
    return;
  }
  status = CCE(hipMemcpy(m_h_genericObjectDevice.m_vertexNormals,
			  m_vertexNormals.data(), dataSize,
			  hipMemcpyHostToDevice));
  if (status != StatusCodes::NoError) {
    return;
  }

  // texture coords ----------------------------------------------------------
  dataSize = m_textureCoords.size()*sizeof(float2);
  status = CCE(hipMalloc((void**)&(m_h_genericObjectDevice.m_textureCoords),
			  dataSize));
  if (status != StatusCodes::NoError) {
    return;
  }
  status = CCE(hipMemcpy(m_h_genericObjectDevice.m_textureCoords,
			  m_textureCoords.data(), dataSize,
			  hipMemcpyHostToDevice));
  if (status != StatusCodes::NoError) {
    return;
  }

  // triangle indices --------------------------------------------------------
  dataSize = m_triangleIndices.size()*sizeof(int);
  status = CCE(hipMalloc((void**)&(m_h_genericObjectDevice.m_triangleIndices),
			  dataSize));
  if (status != StatusCodes::NoError) {
    return;
  }
  status = CCE(hipMemcpy(m_h_genericObjectDevice.m_triangleIndices,
			  m_triangleIndices.data(), dataSize,
			  hipMemcpyHostToDevice));
  if (status != StatusCodes::NoError) {
    return;
  }

  // whole object ------------------------------------------------------------
  status = CCE(hipMemcpy(genericObjectDevice, &m_h_genericObjectDevice,
			  sizeof(GenericObjectDevice), hipMemcpyHostToDevice));
  if (status != StatusCodes::NoError) {
    return;
  }
}

GenericObjectDevice::~GenericObjectDevice()
{
  LOG_TRIVIAL(trace) << "~GenericObjectDevice";
  m_objectType = ObjectType::None;
  m_bmin = make_float3(0.0f, 0.0f, 0.0f);
  m_bmax = make_float3(0.0f, 0.0f, 0.0f);
  // m_material->releaseData();
  // m_material =  = nullptr;
  if (m_scalars) {
    CCE(hipFree(m_scalars));
    m_scalars = nullptr;
  }
  m_numScalars = 0;
  if (m_vectors) {
    CCE(hipFree(m_vectors));
    m_vectors = nullptr;
  }
  m_numVectors = 0;
  if (m_vertices) {
    CCE(hipFree(m_vertices));
    m_vertices = nullptr;
  }
  m_numVertices = 0;
  if (m_vertexColors) {
    CCE(hipFree(m_vertexColors));
    m_vertexColors = nullptr;
  }
  m_numVertexColors = 0;
  if (m_vertexNormals) {
    CCE(hipFree(m_vertexNormals));
    m_vertexNormals = nullptr;
  }
  m_numVertexNormals = 0;
  if (m_textureCoords) {
    CCE(hipFree(m_textureCoords));
    m_textureCoords = nullptr;
  }
  m_numTextureCoords = 0;
  if (m_triangleIndices) {
    CCE(hipFree(m_triangleIndices));
    m_triangleIndices = nullptr;
  }
  m_numTriangleIndices = 0;
}


GenericObjectDevice::GenericObjectDevice(GenericObjectDevice&& other):
  m_objectType(other.m_objectType),
  m_bmin(other.m_bmin),
  m_bmax(other.m_bmax),
  m_material(other.m_material),
  m_scalars(other.m_scalars),
  m_numScalars(other.m_numScalars),
  m_vectors(other.m_vectors),
  m_numVectors(other.m_numVectors),
  m_vertices(other.m_vertices),
  m_numVertices(other.m_numVertices),
  m_vertexColors(other.m_vertexColors),
  m_numVertexColors(other.m_numVertexColors),
  m_vertexNormals(other.m_vertexNormals),
  m_numVertexNormals(other.m_numVertexNormals),
  m_textureCoords(other.m_textureCoords),
  m_numTextureCoords(other.m_numTextureCoords),
  m_triangleIndices(other.m_triangleIndices),
  m_numTriangleIndices(other.m_numTriangleIndices)
{
  other.m_material = nullptr;
  other.m_scalars = nullptr;
  other.m_numScalars = 0;
  other.m_vectors = nullptr;
  other.m_numVectors = 0;
  other.m_vertices = nullptr;
  other.m_numVertices = 0;
  other.m_vertexColors = nullptr;
  other.m_numVertexColors = 0;
  other.m_vertexNormals = nullptr;
  other.m_numVertexNormals = 0;
  other.m_textureCoords = nullptr;
  other.m_numTextureCoords = 0;
  other.m_triangleIndices = nullptr;
  other.m_numTriangleIndices = 0;
}
