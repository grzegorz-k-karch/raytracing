#include "hip/hip_runtime.h"
#include "GenericObject.h"

void GenericObject::copyToDevice(GenericObjectDevice* genericObjectDevice)
{
  GenericObjectDevice *h_genericObject = new GenericObjectDevice;

  // scalars
  int dataSize = m_scalars.size()*sizeof(float);
  CCE(hipMalloc((void**)&(h_genericObject->scalars), dataSize));
  CCE(hipMemcpy(h_genericObject->scalars, m_scalars.data(),
		 dataSize, hipMemcpyHostToDevice));

  // vectors
  dataSize = m_vectors.size()*sizeof(float3);
  CCE(hipMalloc((void**)&(h_genericObject->vectors), dataSize));
  CCE(hipMemcpy(h_genericObject->vectors, m_vectors.data(),
		 dataSize, hipMemcpyHostToDevice));

  // vertices
  dataSize = m_vertices.size()*sizeof(float3);
  CCE(hipMalloc((void**)&(h_genericObject->vertices), dataSize));
  CCE(hipMemcpy(h_genericObject->vertices, m_vertices.data(),
		 dataSize, hipMemcpyHostToDevice));

  // vertex colors
  dataSize = m_vertexColors.size()*sizeof(float3);
  CCE(hipMalloc((void**)&(h_genericObject->vertexColors), dataSize));
  CCE(hipMemcpy(h_genericObject->vertexColors, m_vertexColors.data(),
		 dataSize, hipMemcpyHostToDevice));

  // vertex normals
  dataSize = m_vertexNormals.size()*sizeof(float3);
  CCE(hipMalloc((void**)&(h_genericObject->vertexNormals), dataSize));
  CCE(hipMemcpy(h_genericObject->vertexNormals, m_vertexNormals.data(),
		 dataSize, hipMemcpyHostToDevice));

  // triangle indices
  dataSize = m_triangleIndices.size()*sizeof(int);
  CCE(hipMalloc((void**)&(h_genericObject->triangleIndices), dataSize));
  CCE(hipMemcpy(h_genericObject->triangleIndices, m_triangleIndices.data(),
		 dataSize, hipMemcpyHostToDevice));

  h_genericObject->numScalars = m_scalars.size();
  h_genericObject->numVectors = m_vectors.size();
  h_genericObject->numVertices = m_vertices.size();
  h_genericObject->numVertexColors = m_vertexColors.size();
  h_genericObject->numVertexNormals = m_vertexNormals.size();
  h_genericObject->numTriangleIndices = m_triangleIndices.size();

  // whole object
  CCE(hipMemcpy(genericObjectDevice, h_genericObject,
		 sizeof(GenericObjectDevice), hipMemcpyHostToDevice));

  delete h_genericObject;
}
