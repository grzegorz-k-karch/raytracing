#include "hip/hip_runtime.h"
#include "device_utils.cuh"
#include "GenericObject.h"
#include "GenericMaterial.h"
#include "Camera.cuh"

void copyToDevice(GenericObjectDevice* genericObjectDevice)
{
  GenericObjectDevice *h_genericObject = new GenericObjectDevice;

  int dataSize = m_scalars.size()*sizeof(float);
  hipMalloc((void**)&(h_genericObjects->scalars), dataSize);
  hipMemcpy(h_genericObjects->scalars, m_scalars.data(),
	     dataSize, hipMemcpyHostToDevice);
  
  dataSize = m_vectors.size()*sizeof(float3);
  hipMalloc((void**)&(h_genericObjects->vectors), dataSize);
  hipMemcpy(h_genericObjects->vectors, m_vectors.data(),
	     dataSize, hipMemcpyHostToDevice);

  dataSize = m_vertices.size()*sizeof(float3);
  hipMalloc((void**)&(h_genericObjects->vertices), dataSize);
  hipMemcpy(h_genericObjects->vertices, m_vertices.data(),
	     dataSize, hipMemcpyHostToDevice);

  dataSize = m_vertexColors.size()*sizeof(float3);
  hipMalloc((void**)&(h_genericObjects->vertexColors), dataSize);
  hipMemcpy(h_genericObjects->vertexColors, m_vertexColors.data(),
	     dataSize, hipMemcpyHostToDevice);
  
  dataSize = m_vertexNormals.size()*sizeof(float3);
  hipMalloc((void**)&(h_genericObjects->vertexNormals), dataSize);
  hipMemcpy(h_genericObjects->vertexNormals, m_vertexNormals.data(),
	     dataSize, hipMemcpyHostToDevice);

  dataSize = m_triangleIndices.size()*sizeof(int);
  hipMalloc((void**)&(h_genericObjects->triangleIndices), dataSize);
  hipMemcpy(h_genericObjects->triangleIndices, m_triangleIndices.data(),
	     dataSize, hipMemcpyHostToDevice);

  GenericObjectDevice *d_genericObject;
  hipMalloc((void**)&d_genericObject, sizeof(GenericObjectDevice));
  hipMemcpy(d_genericObject, h_genericObject, sizeof(GenericObjectDevice), hipMemcpyHostToDevice);
  
  delete h_genericObject;
}

